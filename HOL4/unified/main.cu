#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void initialize(int N, float *a, float *b, float *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i] = 0;
		a[i] = 1 + i;
		b[i] = 1 - i;
	}
}

__global__ void addVectors(int N, float *a, float *b, float *c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N){
		c[i] = a[i] + b[i];
	}
}

int main (int argc, char **argv){
	
	if (argc != 2) exit (1);
	int N = atoi(argv[1]);
	int block_size = 512;
	int grid_size = (N-1) / block_size + 1;

	float *a, *b, *c;
	hipMallocManaged (&a, N*sizeof(float));
	hipMallocManaged (&b, N*sizeof(float)); 
	hipMallocManaged (&c, N*sizeof(float));

	initialize<<<grid_size, block_size>>>(N,a,b,c);
	hipDeviceSynchronize();
	addVectors<<<grid_size, block_size>>>(N,a,b,c);
	hipDeviceSynchronize();

	for (int i = 0; i < 5; i++) {
		printf("%f\n", c[i]);
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
